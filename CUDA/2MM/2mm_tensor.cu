#include "hip/hip_runtime.h"
// /***********************************************************************
//  * 2mm_tensor.cu  —  PolyBench “2-Matrix-Multiply” (D = β·D+α·A·B·C)
//  *                   accelerated with NVIDIA Tensor Cores (WMMA).
//  **********************************************************************/
// #define RUN_ON_CPU          // keep CPU reference & comparison
// #include <stdio.h>
// #include <hip/hip_runtime.h>
// #include <hip/hip_fp16.h>
// #include <mma.h>
// using namespace nvcuda;     // wmma::

// #define POLYBENCH_TIME 1
// #include "2mm.cuh"                       // NI,NJ,NK,NL, DATA_TYPE=float
// #include "../../common/polybench.h"
// #include "../../common/polybenchUtilFuncts.h"

// /* ---- WMMA tile shape (Ampere, TF32/FP16) -------------------------- */
// #define WM 16
// #define WN 16
// #define WK 16
// #define WARPS_PER_BLOCK 32               // one warp (32 threads)
// /* ------------------------------------------------------------------- */

// #define GPU_DEVICE 0
// #define ERR_THRESH 0.05f                 // 0.05 % mismatch tolerance

// /*====================================================================*/
// /*------------------------  CPU reference  ---------------------------*/
// static void init_array(int ni,int nj,int nk,int nl,
//                        float *alpha,float *beta,
//                        float *A,float *B,float *C,float *D)
// {
//     *alpha = 32412.0f;
//     *beta  = 2123.0f;
//     for(int i=0;i<ni;i++)
//       for(int k=0;k<nk;k++) A[i*nk+k] = (float)(i*k)/NI;
//     for(int k=0;k<nk;k++)
//       for(int j=0;j<nj;j++) B[k*nj+j] = (float)(k*(j+1))/NJ;
//     for(int l=0;l<nl;l++)
//       for(int j=0;j<nj;j++) C[l*nj+j] = (float)(l*(j+3))/NL;
//     for(int i=0;i<ni;i++)
//       for(int l=0;l<nl;l++) D[i*nl+l] = (float)(i*(l+2))/NK;
// }

// static void mm2_cpu(int ni,int nj,int nk,int nl,
//                     float alpha,float beta,
//                     const float *A,const float *B,const float *C,
//                     float *D)
// {
//     float *tmp = (float*)malloc(sizeof(float)*ni*nj);
//     for(int i=0;i<ni;i++)
//       for(int j=0;j<nj;j++){
//         float acc=0.f;
//         for(int k=0;k<nk;k++) acc += alpha*A[i*nk+k]*B[k*nj+j];
//         tmp[i*nj+j]=acc;
//       }

//     for(int i=0;i<ni;i++)
//       for(int l=0;l<nl;l++){
//         float acc=beta*D[i*nl+l];
//         for(int j=0;j<nj;j++) acc+=tmp[i*nj+j]*C[j*nl+l];
//         D[i*nl+l]=acc;
//       }
//     free(tmp);
// }

// static void compareResults(int ni,int nl,const float *D,const float *Dg)
// {
//     int fail=0;
//     for(int i=0;i<ni;i++)
//       for(int l=0;l<nl;l++)
//         if(percentDiff(D[i*nl+l],Dg[i*nl+l])>ERR_THRESH) ++fail;
//     printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold "
//            "(%4.2f%%): %d\n", ERR_THRESH, fail);
// }

// static void GPU_argv_init()
// {
//     hipDeviceProp_t prop; hipGetDeviceProperties(&prop,GPU_DEVICE);
//     printf("setting device %d with name %s\n",GPU_DEVICE,prop.name);
//     hipSetDevice(GPU_DEVICE);
// }

// /*====================================================================*/
// /*-----------------------  Device helpers  ---------------------------*/
// __global__ void f32_to_f16(const float *in,__half *out,int n)
// {
//     int idx=blockIdx.x*blockDim.x+threadIdx.x;
//     if(idx<n) out[idx]=__float2half(in[idx]);
// }

// /*---------------- Tensor-Core GEMM #1  (tmp=α·A·B) ------------------*/
// __global__ void wmma_gemm1(int M,int N,int K,
//                            const __half *A_h,const __half *B_h,
//                            float alpha,float *tmp_f)
// {
//     int tm=blockIdx.y, tn=blockIdx.x;
//     wmma::fragment<wmma::accumulator,WM,WN,WK,float> acc;
//     wmma::fill_fragment(acc,0.0f);
//     for(int k=0;k<K;k+=WK){
//         wmma::fragment<wmma::matrix_a,WM,WN,WK,__half,wmma::row_major> Af;
//         wmma::fragment<wmma::matrix_b,WM,WN,WK,__half,wmma::row_major> Bf;
//         wmma::load_matrix_sync(Af,A_h+(tm*WM)*K+k,K);
//         wmma::load_matrix_sync(Bf,B_h+k*N+tn*WN,N);
//         wmma::mma_sync(acc,Af,Bf,acc);
//     }
//     for(int i=0;i<acc.num_elements;i++) acc.x[i]*=alpha;
//     wmma::store_matrix_sync(tmp_f+(tm*WM)*N+tn*WN,acc,N,wmma::mem_row_major);
// }

// /*---------------- Tensor-Core GEMM #2  (D=β·D+tmp·C) ----------------*/
// __global__ void wmma_gemm2(int M,int N,int K,
//                            const __half *tmp_h,const __half *C_h,
//                            float beta,float *D_f)
// {
//     int tm=blockIdx.y, tn=blockIdx.x;
//     wmma::fragment<wmma::accumulator,WM,WN,WK,float> acc;
//     wmma::fill_fragment(acc,0.0f);
//     for(int k=0;k<K;k+=WK){
//         wmma::fragment<wmma::matrix_a,WM,WN,WK,__half,wmma::row_major> Af;
//         wmma::fragment<wmma::matrix_b,WM,WN,WK,__half,wmma::row_major> Bf;
//         wmma::load_matrix_sync(Af,tmp_h+(tm*WM)*K+k,K);
//         wmma::load_matrix_sync(Bf,C_h  +k*N +tn*WN,N);
//         wmma::mma_sync(acc,Af,Bf,acc);
//     }
//     wmma::fragment<wmma::accumulator,WM,WN,WK,float> old;
//     float *dst=D_f+(tm*WM)*N+tn*WN;
//     wmma::load_matrix_sync(old,dst,N,wmma::mem_row_major);
//     for(int i=0;i<acc.num_elements;i++) acc.x[i]+=beta*old.x[i];
//     wmma::store_matrix_sync(dst,acc,N,wmma::mem_row_major);
// }

// /*====================================================================*/
// /*-------------------------  GPU driver  -----------------------------*/
// void mm2Cuda(int ni,int nj,int nk,int nl,
//              float alpha,float beta,
//              const float *A,const float *B,const float *C,
//              const float *D_in,float *D_out)
// {
//     size_t sA=sizeof(float)*ni*nk, sB=sizeof(float)*nk*nj,
//            sC=sizeof(float)*nl*nj, sD=sizeof(float)*ni*nl,
//            sT=sizeof(float)*ni*nj;

//     /* FP32 device buffers */
//     float *dA_f,*dB_f,*dC_f,*dD_f,*dTmp_f;
//     hipMalloc(&dA_f,sA); hipMalloc(&dB_f,sB);
//     hipMalloc(&dC_f,sC); hipMalloc(&dD_f,sD);
//     hipMalloc(&dTmp_f,sT);
//     hipMemcpy(dA_f,A,sA,hipMemcpyHostToDevice);
//     hipMemcpy(dB_f,B,sB,hipMemcpyHostToDevice);
//     hipMemcpy(dC_f,C,sC,hipMemcpyHostToDevice);
//     hipMemcpy(dD_f,D_in,sD,hipMemcpyHostToDevice);

//     /* FP16 copies */
//     __half *dA_h,*dB_h,*dC_h,*dTmp_h;
//     hipMalloc(&dA_h,sA/2); hipMalloc(&dB_h,sB/2);
//     hipMalloc(&dC_h,sC/2); hipMalloc(&dTmp_h,sT/2);

//     int threads=256;
//     f32_to_f16<<<(ni*nk+threads-1)/threads,threads>>>(dA_f,dA_h,ni*nk);
//     f32_to_f16<<<(nk*nj+threads-1)/threads,threads>>>(dB_f,dB_h,nk*nj);
//     f32_to_f16<<<(nl*nj+threads-1)/threads,threads>>>(dC_f,dC_h,nl*nj);

//     dim3 block(WARPS_PER_BLOCK,1);
//     dim3 grid1(nj/WM, ni/WM);          // tmp = αAB
//     dim3 grid2(nl/WM, ni/WM);          // D   = βD+tmpC

//     polybench_start_instruments;

//     wmma_gemm1<<<grid1,block>>>(ni,nj,nk,dA_h,dB_h,alpha,dTmp_f);
//     f32_to_f16<<<(ni*nj+threads-1)/threads,threads>>>(dTmp_f,dTmp_h,ni*nj);
//     wmma_gemm2<<<grid2,block>>>(ni,nl,nj,dTmp_h,dC_h,beta,dD_f);

//     hipDeviceSynchronize();
//     printf("GPU Time in seconds:\n");
//     polybench_stop_instruments;
//     polybench_print_instruments;

//     hipMemcpy(D_out,dD_f,sD,hipMemcpyDeviceToHost);
//     hipFree(dA_f); hipFree(dB_f); hipFree(dC_f);
//     hipFree(dD_f); hipFree(dTmp_f);
//     hipFree(dA_h); hipFree(dB_h); hipFree(dC_h); hipFree(dTmp_h);
// }

// /*====================================================================*/
// /*------------------------------ main --------------------------------*/
// int main()
// {
//     /* flat row-major host matrices */
//     float *A  =(float*)malloc(sizeof(float)*NI*NK);
//     float *B  =(float*)malloc(sizeof(float)*NK*NJ);
//     float *C  =(float*)malloc(sizeof(float)*NL*NJ);
//     float *D  =(float*)malloc(sizeof(float)*NI*NL);
//     float *Dg =(float*)malloc(sizeof(float)*NI*NL);

//     float alpha,beta;
//     init_array(NI,NJ,NK,NL,&alpha,&beta,A,B,C,D);
//     GPU_argv_init();

//     mm2Cuda(NI,NJ,NK,NL,alpha,beta,A,B,C,D,Dg);

// #ifdef RUN_ON_CPU
//     polybench_start_instruments;
//     mm2_cpu(NI,NJ,NK,NL,alpha,beta,A,B,C,D);
//     printf("CPU Time in seconds:\n");
//     polybench_stop_instruments;
//     polybench_print_instruments;
//     compareResults(NI,NL,D,Dg);
// #endif

//     free(A); free(B); free(C); free(D); free(Dg);
//     return 0;
// }

// /* bring in PolyBench timer implementation */
// #include "../../common/polybench.c"


/*************************************************************************
 * PolyBench/GPU 2-MM — Tensor-Core version
 *   Computes D = β·D + α·A·B·C   (all row-major)
 *   Only GPU math is changed; everything else is identical to
 *   the original shared-memory–tiled program.
 *************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>
using namespace nvcuda;                    // wmma::

#define POLYBENCH_TIME 1
#include "2mm.cuh"                         // NI,NJ,NK,NL, DATA_TYPE=float
#include "../../common/polybench.h"
#include "../../common/polybenchUtilFuncts.h"

/* ---------------- Tensor-Core WMMA tile ---------------------------- */
#define WM 16
#define WN 16
#define WK 16                  /* 16×16×16 MMA                        */
#define WARPS_PER_BLOCK 32     /* one warp / WMMA tile               */
/* ------------------------------------------------------------------- */

#define GPU_DEVICE 0
#define ERR_THRESH 0.05f       /* 0.05 % max relative-error allowed   */

/* =================================================================== */
/* ----------  ORIGINAL helper functions (unchanged)  ---------------- */
static void init_array(int ni,int nj,int nk,int nl,
                    DATA_TYPE *alpha, DATA_TYPE *beta,
                    DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk),
                    DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj),
                    DATA_TYPE POLYBENCH_2D(C,NL,NJ,nl,nj),
                    DATA_TYPE POLYBENCH_2D(D,NI,NL,ni,nl))
{
    *alpha = 32412.0f;
    *beta  = 2123.0f;
    for(int i=0;i<ni;i++)
    for(int k=0;k<nk;k++) A[i][k] = (float)(i*k)/NI;
    for(int k=0;k<nk;k++)
    for(int j=0;j<nj;j++) B[k][j] = (float)(k*(j+1))/NJ;
    for(int l=0;l<nl;l++)
    for(int j=0;j<nj;j++) C[l][j] = (float)(l*(j+3))/NL;
    for(int i=0;i<ni;i++)
    for(int l=0;l<nl;l++) D[i][l] = (float)(i*(l+2))/NK;
}

static void compareResults(int ni,int nl,
                        DATA_TYPE POLYBENCH_2D(D,NI,NL,ni,nl),
                        DATA_TYPE POLYBENCH_2D(Dg,NI,NL,ni,nl))
{
    int fail=0;
    for(int i=0;i<ni;i++)
    for(int l=0;l<nl;l++)
        if(percentDiff(D[i][l],Dg[i][l])>ERR_THRESH) ++fail;
    printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold "
        "(%4.2f%%): %d\n", ERR_THRESH, fail);
}

static void GPU_argv_init()
{
    hipDeviceProp_t p; hipGetDeviceProperties(&p,GPU_DEVICE);
    printf("setting device %d with name %s\n",GPU_DEVICE,p.name);
    hipSetDevice(GPU_DEVICE);
}

/* -------------------  optional CPU reference  --------------------- */
static void mm2_cpu(int ni,int nj,int nk,int nl,
                    DATA_TYPE alpha, DATA_TYPE beta,
                    DATA_TYPE POLYBENCH_2D(tmp,NI,NJ,ni,nj),
                    DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk),
                    DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj),
                    DATA_TYPE POLYBENCH_2D(C,NL,NJ,nl,nj),
                    DATA_TYPE POLYBENCH_2D(D,NI,NL,ni,nl))
{
    for(int i=0;i<ni;i++)
    for(int j=0;j<nj;j++){
        DATA_TYPE acc=0.f;
        for(int k=0;k<nk;k++) acc+=alpha*A[i][k]*B[k][j];
        tmp[i][j]=acc;
    }
    for(int i=0;i<ni;i++)
    for(int l=0;l<nl;l++){
        DATA_TYPE acc=beta*D[i][l];
        for(int j=0;j<nj;j++) acc+=tmp[i][j]*C[j][l];
        D[i][l]=acc;
    }
}

/* =================================================================== */
/* ---------------------  device helper kernels  --------------------- */
__global__ void f32_to_f16(const float *in,__half *out,int n)
{
    int idx=blockIdx.x*blockDim.x+threadIdx.x;
    if(idx<n) out[idx]=__float2half(in[idx]);
}

/* tmp = α · A · B   (M=NI, N=NJ, K=NK) */
__global__ void wmma_kernel1(int M,int N,int K,float alpha,
                            const __half *A,const __half *B,float *tmp)
{
    int tm=blockIdx.y, tn=blockIdx.x;

    wmma::fragment<wmma::accumulator,WM,WN,WK,float> acc;
    wmma::fill_fragment(acc,0.0f);

    for(int k=0;k<K;k+=WK){
        wmma::fragment<wmma::matrix_a,WM,WN,WK,__half,wmma::row_major> Af;
        wmma::fragment<wmma::matrix_b,WM,WN,WK,__half,wmma::row_major> Bf;
        wmma::load_matrix_sync(Af, A+(tm*WM)*K+k, K);
        wmma::load_matrix_sync(Bf, B+k*N+tn*WN, N);
        wmma::mma_sync(acc,Af,Bf,acc);
    }
    for(int i=0;i<acc.num_elements;i++) acc.x[i]*=alpha;
    wmma::store_matrix_sync(tmp+(tm*WM)*N+tn*WN,acc,N,wmma::mem_row_major);
}

/* D = β·D + tmp · C   (M=NI, N=NL, K=NJ) */
__global__ void wmma_kernel2(int M,int N,int K,float beta,
                            const __half *tmp_h,const __half *C_h,float *D)
{
    int tm=blockIdx.y, tn=blockIdx.x;

    wmma::fragment<wmma::accumulator,WM,WN,WK,float> acc;
    wmma::fill_fragment(acc,0.0f);

    for(int k=0;k<K;k+=WK){
        wmma::fragment<wmma::matrix_a,WM,WN,WK,__half,wmma::row_major> Af;
        wmma::fragment<wmma::matrix_b,WM,WN,WK,__half,wmma::row_major> Bf;
        wmma::load_matrix_sync(Af,tmp_h+(tm*WM)*K+k,K);
        wmma::load_matrix_sync(Bf,C_h  +k*N +tn*WN,N);
        wmma::mma_sync(acc,Af,Bf,acc);
    }
    /* β scaling with original D */
    wmma::fragment<wmma::accumulator,WM,WN,WK,float> Dold;
    float *dst=D+(tm*WM)*N+tn*WN;
    wmma::load_matrix_sync(Dold,dst,N,wmma::mem_row_major);
    for(int i=0;i<acc.num_elements;i++) acc.x[i]+=beta*Dold.x[i];
    wmma::store_matrix_sync(dst,acc,N,wmma::mem_row_major);
}

/* =================================================================== */
/* -----------------------  GPU driver  ------------------------------ */
void mm2Cuda(int ni,int nj,int nk,int nl,
            DATA_TYPE alpha, DATA_TYPE beta,
            DATA_TYPE POLYBENCH_2D(tmp,NI,NJ,ni,nj),
            DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk),
            DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj),
            DATA_TYPE POLYBENCH_2D(C,NL,NJ,nl,nj),
            DATA_TYPE POLYBENCH_2D(D,NI,NL,ni,nl),
            DATA_TYPE POLYBENCH_2D(Dg,NI,NL,ni,nl))
{
    size_t sA=sizeof(float)*NI*NK, sB=sizeof(float)*NK*NJ,
        sC=sizeof(float)*NL*NJ, sD=sizeof(float)*NI*NL,
        sT=sizeof(float)*NI*NJ;

    /* FP32 device buffers */
    float *dA,*dB,*dC,*dD,*dTmp;
    hipMalloc(&dA,sA); hipMalloc(&dB,sB);
    hipMalloc(&dC,sC); hipMalloc(&dD,sD); hipMalloc(&dTmp,sT);
    hipMemcpy(dA,&A[0][0],sA,hipMemcpyHostToDevice);
    hipMemcpy(dB,&B[0][0],sB,hipMemcpyHostToDevice);
    hipMemcpy(dC,&C[0][0],sC,hipMemcpyHostToDevice);
    hipMemcpy(dD,&D[0][0],sD,hipMemcpyHostToDevice);

    /* FP16 copies for WMMA */
    __half *dA_h,*dB_h,*dC_h,*dTmp_h;
    hipMalloc(&dA_h,sA/2); hipMalloc(&dB_h,sB/2);
    hipMalloc(&dC_h,sC/2); hipMalloc(&dTmp_h,sT/2);

    int threads=256;
    f32_to_f16<<<(NI*NK+threads-1)/threads,threads>>>(dA,dA_h,NI*NK);
    f32_to_f16<<<(NK*NJ+threads-1)/threads,threads>>>(dB,dB_h,NK*NJ);
    f32_to_f16<<<(NL*NJ+threads-1)/threads,threads>>>(dC,dC_h,NL*NJ);

    dim3 block(WARPS_PER_BLOCK,1);
    dim3 grid1(NJ/WM, NI/WM);          /* tmp = αAB   */
    dim3 grid2(NL/WM, NI/WM);          /* D = βD+tmpC */

    polybench_start_instruments;

    wmma_kernel1<<<grid1,block>>>(NI,NJ,NK,alpha,dA_h,dB_h,dTmp);
    f32_to_f16<<<(NI*NJ+threads-1)/threads,threads>>>(dTmp,dTmp_h,NI*NJ);
    wmma_kernel2<<<grid2,block>>>(NI,NL,NJ,beta,dTmp_h,dC_h,dD);

    hipDeviceSynchronize();
    printf("GPU Time in seconds:\n");
    polybench_stop_instruments;
    polybench_print_instruments;

    hipMemcpy(&Dg[0][0],dD,sD,hipMemcpyDeviceToHost);

    hipFree(dA); hipFree(dB); hipFree(dC); hipFree(dD); hipFree(dTmp);
    hipFree(dA_h); hipFree(dB_h); hipFree(dC_h); hipFree(dTmp_h);
}

/* =================================================================== */
/* ------------------------------ main ------------------------------- */
int main()
{
    int ni=NI,nj=NJ,nk=NK,nl=NL;
    DATA_TYPE alpha,beta;

    POLYBENCH_2D_ARRAY_DECL(tmp ,DATA_TYPE,NI,NJ,ni,nj);
    POLYBENCH_2D_ARRAY_DECL(A   ,DATA_TYPE,NI,NK,ni,nk);
    POLYBENCH_2D_ARRAY_DECL(B   ,DATA_TYPE,NK,NJ,nk,nj);
    POLYBENCH_2D_ARRAY_DECL(C   ,DATA_TYPE,NL,NJ,nl,nj);
    POLYBENCH_2D_ARRAY_DECL(D   ,DATA_TYPE,NI,NL,ni,nl);
    POLYBENCH_2D_ARRAY_DECL(Dgpu,DATA_TYPE,NI,NL,ni,nl);

    init_array(ni,nj,nk,nl,&alpha,&beta,
            POLYBENCH_ARRAY(A),POLYBENCH_ARRAY(B),
            POLYBENCH_ARRAY(C),POLYBENCH_ARRAY(D));
    GPU_argv_init();

    mm2Cuda(ni,nj,nk,nl,alpha,beta,
            POLYBENCH_ARRAY(tmp),POLYBENCH_ARRAY(A),POLYBENCH_ARRAY(B),
            POLYBENCH_ARRAY(C),  POLYBENCH_ARRAY(D), POLYBENCH_ARRAY(Dgpu));


    polybench_start_instruments;
    mm2_cpu(ni,nj,nk,nl,alpha,beta,
            POLYBENCH_ARRAY(tmp),POLYBENCH_ARRAY(A),POLYBENCH_ARRAY(B),
            POLYBENCH_ARRAY(C),  POLYBENCH_ARRAY(D));
    printf("CPU Time in seconds:\n");
    polybench_stop_instruments;
    polybench_print_instruments;
    compareResults(ni,nl,POLYBENCH_ARRAY(D),POLYBENCH_ARRAY(Dgpu));

    POLYBENCH_FREE_ARRAY(tmp); POLYBENCH_FREE_ARRAY(A); POLYBENCH_FREE_ARRAY(B);
    POLYBENCH_FREE_ARRAY(C);   POLYBENCH_FREE_ARRAY(D); POLYBENCH_FREE_ARRAY(Dgpu);
    return 0;
}

/* bring in PolyBench timer implementation */
#include "../../common/polybench.c"
 