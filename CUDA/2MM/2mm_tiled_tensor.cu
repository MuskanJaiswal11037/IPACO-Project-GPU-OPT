#include "hip/hip_runtime.h"
/**
 * 2mm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

 #include <stdio.h>
 #include <stdlib.h>
 #include <math.h>
 #include <assert.h>
 #include <unistd.h>
 #include <sys/time.h>
 #include <hip/hip_runtime.h>

 #include <hipblas.h>          // add after #include <hip/hip_runtime.h>

inline void gpuAssert(hipblasStatus_t stat, const char *file, int line)
{
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS error %d at %s:%d\n", stat, file, line);
        exit(EXIT_FAILURE);
    }
}
#define CUBLAS_CHECK(stmt) gpuAssert((stmt), __FILE__, __LINE__)

 
 #define POLYBENCH_TIME 1
 
 #include "2mm.cuh"
 #include "../../common/polybench.h"
 #include "../../common/polybenchUtilFuncts.h"
 
 #define TILE_DIM 16            /* 16×16 = 256 threads per block     */
 #define DIM_THREAD_BLOCK_X TILE_DIM
 #define DIM_THREAD_BLOCK_Y TILE_DIM
 
 //define the error threshold for the results "not matching"
 #define PERCENT_DIFF_ERROR_THRESHOLD 0.05
 
 #define GPU_DEVICE 0
 
 #define RUN_ON_CPU
 
 
 void init_array(int ni, int nj, int nk, int nl, DATA_TYPE *alpha, DATA_TYPE *beta, DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk), 
         DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj), DATA_TYPE POLYBENCH_2D(C, NL, NJ, nl, nj), 
         DATA_TYPE POLYBENCH_2D(D, NI, NL, ni, nl))
 {
     int i, j;
 
     *alpha = 32412;
     *beta = 2123;
 
     for (i = 0; i < ni; i++)
     {
         for (j = 0; j < nk; j++)
         {
             A[i][j] = ((DATA_TYPE) i*j) / NI;
         }
     }
 
     for (i = 0; i < nk; i++)
     {
         for (j = 0; j < nj; j++)
         {
             B[i][j] = ((DATA_TYPE) i*(j+1)) / NJ;
         }
     }
 
     for (i = 0; i < nl; i++)
     {
         for (j = 0; j < nj; j++)
         {
             C[i][j] = ((DATA_TYPE) i*(j+3)) / NL;
         }
     }
 
     for (i = 0; i < ni; i++)
     {
         for (j = 0; j < nl; j++)
         {
             D[i][j] = ((DATA_TYPE) i*(j+2)) / NK;	
         }
     }
 }
 
 
 void compareResults(int ni, int nl, DATA_TYPE POLYBENCH_2D(D, NI, NL, ni, nl), DATA_TYPE POLYBENCH_2D(D_outputFromGpu, NI, NL, ni, nl))
 {
     int i,j,fail;
     fail = 0;
 
     for (i=0; i < ni; i++)
     {
         for (j=0; j < nl; j++)
         {
             if (percentDiff(D[i][j], D_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD)
             {
                 fail++;
             }
         }
     }
     
     // print results
     printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
 }
 
 
 void GPU_argv_init()
 {
     hipDeviceProp_t deviceProp;
     hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
     printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
     hipSetDevice( GPU_DEVICE );
 }
 
 __global__ void mm2_kernel1(int ni, int nj, int nk, int nl, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE *tmp, const DATA_TYPE *A, const DATA_TYPE *B){
     __shared__ DATA_TYPE As[TILE_DIM][TILE_DIM];
     __shared__ DATA_TYPE Bs[TILE_DIM][TILE_DIM];
 
    int row = blockIdx.y * TILE_DIM + threadIdx.y;   // i
    int col = blockIdx.x * TILE_DIM + threadIdx.x;   // j
 
     DATA_TYPE acc = 0.0;
 
     /* loop over tiles of the K dimension */
     for (int t = 0; t < (nk + TILE_DIM - 1) / TILE_DIM; ++t) {
 
     /* global indices of the element this thread will copy */
     int tiled_k = t * TILE_DIM + threadIdx.x;     // k for As
     int tiled_kT = t * TILE_DIM + threadIdx.y;    // k for Bs
 
     /* load one element of A and B into shared memory (with bounds check) */
     As[threadIdx.y][threadIdx.x] =
     (row < ni && tiled_k < nk) ?
     A[row * nk + tiled_k] : 0.0;
 
     Bs[threadIdx.y][threadIdx.x] =
     (col < nj && tiled_kT < nk) ?
     B[tiled_kT * nj + col] : 0.0;
 
     __syncthreads();
 
     /* dot product for this tile */
     #pragma unroll
     for (int k = 0; k < TILE_DIM; ++k)
     acc += As[threadIdx.y][k] * Bs[k][threadIdx.x];
 
     __syncthreads();
     }
 
     if (row < ni && col < nj)
     tmp[row * nj + col] = alpha * acc;   // multiply α once
 }
 
 
 __global__ void mm2_kernel2(int ni, int nj, int nk, int nl, DATA_TYPE alpha, DATA_TYPE beta, const DATA_TYPE *tmp, const DATA_TYPE *C, DATA_TYPE *D){
     __shared__ DATA_TYPE Ts[TILE_DIM][TILE_DIM];
     __shared__ DATA_TYPE Cs[TILE_DIM][TILE_DIM];
 
     int row = blockIdx.y * TILE_DIM + threadIdx.y;   // i
     int col = blockIdx.x * TILE_DIM + threadIdx.x;   // j
 
     DATA_TYPE acc = 0.0;
 
     for (int t = 0; t < (nj + TILE_DIM - 1) / TILE_DIM; ++t) {
 
     int tiled_k = t * TILE_DIM + threadIdx.x;     // k for Ts
     int tiled_kT = t * TILE_DIM + threadIdx.y;    // k for Cs
 
     Ts[threadIdx.y][threadIdx.x] =
     (row < ni && tiled_k < nj) ?
     tmp[row * nj + tiled_k] : 0.0;
 
     Cs[threadIdx.y][threadIdx.x] =
     (tiled_kT < nj && col < nl) ?
     C[tiled_kT * nl + col] : 0.0;
 
     __syncthreads();
 
     #pragma unroll
     for (int k = 0; k < TILE_DIM; ++k)
     acc += Ts[threadIdx.y][k] * Cs[k][threadIdx.x];
 
     __syncthreads();
     }
 
     if (row < ni && col < nl)
     D[row * nl + col] = beta * D[row * nl + col] + acc;
 }
 
 
 void mm2_cpu(int ni, int nj, int nk, int nl,
         DATA_TYPE alpha,
         DATA_TYPE beta,
         DATA_TYPE POLYBENCH_2D(tmp,NI,NJ,ni,nj),
         DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk),
         DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj),
         DATA_TYPE POLYBENCH_2D(C,NL,NJ,nl,nj),
         DATA_TYPE POLYBENCH_2D(D,NI,NL,ni,nl))
 {
     int i, j, k;
     
     /* D := alpha*A*B*C + beta*D */
     for (i = 0; i < _PB_NI; i++)
     {
         for (j = 0; j < _PB_NJ; j++)
         {
             tmp[i][j] = 0;
             for (k = 0; k < _PB_NK; ++k)
             {
                 tmp[i][j] += alpha * A[i][k] * B[k][j];
             }
         }
     }
 
     for (i = 0; i < _PB_NI; i++)
     {
         for (j = 0; j < _PB_NL; j++)
         {
             D[i][j] *= beta;
             for (k = 0; k < _PB_NJ; ++k)
             {
                 D[i][j] += tmp[i][k] * C[k][j];
             }
         }
     }
 }

 /* DCE code. Must scan the entire live-out data.
    Can be used also to check the correctness of the output. */
 static
 void print_array(int ni, int nl,
          DATA_TYPE POLYBENCH_2D(D,NI,NL,ni,nl))
 {
   int i, j;
 
   for (i = 0; i < ni; i++)
     for (j = 0; j < nl; j++) {
     fprintf (stderr, DATA_PRINTF_MODIFIER, D[i][j]);
     if ((i * ni + j) % 20 == 0) fprintf (stderr, "\n");
     }
   fprintf (stderr, "\n");
 }
 
 
 void mm2Cuda(int ni, int nj, int nk, int nl, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(tmp,NI,NJ,ni,nj), 
     DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk), DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj), DATA_TYPE POLYBENCH_2D(C,NL,NJ,nl,nj), 
     DATA_TYPE POLYBENCH_2D(D,NI,NL,ni,nl), DATA_TYPE POLYBENCH_2D(D_outputFromGpu,NI,NL,ni,nl))
 {
     DATA_TYPE *tmp_gpu;
     DATA_TYPE *A_gpu;
     DATA_TYPE *B_gpu;
     DATA_TYPE *C_gpu;
     DATA_TYPE *D_gpu;
 
     hipMalloc((void **)&tmp_gpu, sizeof(DATA_TYPE) * NI * NJ);
     hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NK);
     hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NK * NJ);
     hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NL * NJ);
     hipMalloc((void **)&D_gpu, sizeof(DATA_TYPE) * NI * NL);
     
     hipMemcpy(tmp_gpu, tmp, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
     hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
     hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
     hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * NL * NJ, hipMemcpyHostToDevice);
     hipMemcpy(D_gpu, D, sizeof(DATA_TYPE) * NI * NL, hipMemcpyHostToDevice);	
         
     dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
     dim3 grid1((size_t)ceil( ((float)NJ) / ((float)block.x) ), (size_t)ceil( ((float)NI) / ((float)block.y)) );
     dim3 grid2((size_t)ceil( ((float)NL) / ((float)block.x) ), (size_t)ceil( ((float)NI) / ((float)block.y)) );
 

     hipblasHandle_t handle;
    CUBLAS_CHECK( hipblasCreate(&handle) );
    CUBLAS_CHECK( hipblasSetMathMode(handle,
                                    HIPBLAS_TF32_TENSOR_OP_MATH) );

    const float  f_alpha = alpha;
    const float  f_beta0 = 0.0f;
    const float  f_beta  = beta;

    /* 2.b  tmp = alpha * A·B            (row-major => use transposes) */
    // cuBLAS is column-major.  The trick is to compute tmp^T = B^T·A^T
    CUBLAS_CHECK(
      hipblasSgemm(handle,
                  HIPBLAS_OP_T, HIPBLAS_OP_T,          // B^T · A^T
                  NJ,  NI,  NK,                      // m, n, k   (note: swapped)
                  &f_alpha,
                  B_gpu, NK,                         // B^T (lda = NK)
                  A_gpu, NI,                         // A^T (ldb = NI)
                  &f_beta0,
                  tmp_gpu, NJ));                     // tmp^T (ldc = NJ)

    /* 2.c  D   = beta·D + tmp·C         (again as transposed product) */
    // We already have tmp^T in tmp_gpu.  Compute  D^T = C^T · tmp^T
    CUBLAS_CHECK(
      hipblasSgemm(handle,
                  HIPBLAS_OP_T, HIPBLAS_OP_N,          // C^T · tmp^T
                  NL,  NI,  NJ,                      // m, n, k
                  &f_alpha,
                  C_gpu, NJ,                         // C^T
                  tmp_gpu, NJ,                       // tmp^T
                  &f_beta,
                  D_gpu, NL));                       // D^T  (in-place)

    

     /* Start timer. */
       polybench_start_instruments;
 
     mm2_kernel1<<<grid1,block>>>(ni, nj, nk, nl, alpha, beta, tmp_gpu, A_gpu, B_gpu);
     hipDeviceSynchronize();
     mm2_kernel2<<<grid2,block>>>(ni, nj, nk, nl, alpha, beta, tmp_gpu, C_gpu, D_gpu);
     hipDeviceSynchronize();
 
     printf("GPU Time in seconds:\n");
       polybench_stop_instruments;
      polybench_print_instruments;
 
     hipMemcpy(D_outputFromGpu, D_gpu, sizeof(DATA_TYPE) * NI * NL, hipMemcpyDeviceToHost);
 
     hipFree(tmp_gpu);
     hipFree(A_gpu);
     hipFree(B_gpu);
     hipFree(C_gpu);
     hipFree(D_gpu);

     CUBLAS_CHECK( hipblasDestroy(handle) );
 }
 
 int main(int argc, char** argv)
 {
     /* Retrieve problem size. */
     int ni = NI;
     int nj = NJ;
     int nk = NK;
     int nl = NL;
 
     /* Variable declaration/allocation. */
     DATA_TYPE alpha;
     DATA_TYPE beta;
     POLYBENCH_2D_ARRAY_DECL(tmp,DATA_TYPE,NI,NJ,ni,nj);
     POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,NI,NK,ni,nk);
     POLYBENCH_2D_ARRAY_DECL(B,DATA_TYPE,NK,NJ,nk,nj);
     POLYBENCH_2D_ARRAY_DECL(C,DATA_TYPE,NL,NJ,nl,nj);
     POLYBENCH_2D_ARRAY_DECL(D,DATA_TYPE,NI,NL,ni,nl);
     POLYBENCH_2D_ARRAY_DECL(D_outputFromGpu,DATA_TYPE,NI,NL,ni,nl);
     
     /* Initialize array(s). */
       init_array(ni, nj, nk, nl, &alpha, &beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(D));
     GPU_argv_init();
 
     mm2Cuda(ni, nj, nk, nl, alpha, beta, POLYBENCH_ARRAY(tmp), POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), 
         POLYBENCH_ARRAY(D), POLYBENCH_ARRAY(D_outputFromGpu));
 
     #ifdef RUN_ON_CPU
 
         /* Start timer. */
           polybench_start_instruments;
 
         mm2_cpu(ni, nj, nk, nl, alpha, beta, POLYBENCH_ARRAY(tmp), POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(D));
 
         printf("CPU Time in seconds:\n");
           polybench_stop_instruments;
          polybench_print_instruments;
 
         compareResults(ni, nl, POLYBENCH_ARRAY(D), POLYBENCH_ARRAY(D_outputFromGpu));
 
     #else //print output to stderr so no dead code elimination
 
         print_array(ni, nl, POLYBENCH_ARRAY(D_outputFromGpu));
 
     #endif //RUN_ON_CPU
 
     POLYBENCH_FREE_ARRAY(tmp);
     POLYBENCH_FREE_ARRAY(A);
     POLYBENCH_FREE_ARRAY(B);
     POLYBENCH_FREE_ARRAY(C);
     POLYBENCH_FREE_ARRAY(D);
     POLYBENCH_FREE_ARRAY(D_outputFromGpu);
 
       return 0;
 }
 
 #include "../../common/polybench.c"
 